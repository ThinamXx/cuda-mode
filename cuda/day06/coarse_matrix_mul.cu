#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define COARSE_FACTOR 4

void printMatrix(float *matrix, int width, int height) {
    printf("\n\n");

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            printf("%f ", matrix[i * width + j]);
        }
        printf("\n");
    }

    printf("\n\n");
}

__host__ int calculate_appropriate_tile_size(hipDeviceProp_t device_prop) {
    // Refer this implementation: ./dynamic_matrix_mul.cu

    // 1. Shared memory constraints. 
    size_t shared_mem_per_block = device_prop.sharedMemPerBlock;
    size_t max_tile_elements = shared_mem_per_block / (2 * sizeof(float)); // 2 tiles for A and B.
    int tile_size_from_shared = (int)floor(sqrt(max_tile_elements));

    // 2. Thread count constraints. 
    int max_threads_per_block = device_prop.maxThreadsPerBlock;
    int tile_size_from_threads = (int)floor(sqrt(max_threads_per_block));

    // 3. Warp size constraints. 
    int warp_size = device_prop.warpSize;
    
    int tile_size = min(min(tile_size_from_shared, tile_size_from_threads), warp_size);

    // Ensure the tile size is a multiple of the warp size.
    tile_size = (tile_size / warp_size) * warp_size;

    // Print the GPU properties and the tile size.
    printf("Device name: %s\n", device_prop.name);
    printf("Shared memory per block: %zu bytes\n", shared_mem_per_block);
    printf("Max threads per block: %d\n", max_threads_per_block);
    printf("Warp size: %d\n", warp_size);
    printf("Tile size: %d\n", tile_size);
    printf("\n");

    return tile_size;
}

__global__ void coarseMatrixMul_kernel(
    int J, 
    int K, 
    int L, 
    float *A, 
    float *B, 
    float *C,
    int tile_size, 
    unsigned A_tile_offset, 
    unsigned B_tile_offset
) {
    extern __shared__ float A_tile_B_tile[];

    float *A_tile = (float *)(A_tile_B_tile);
    float *B_tile = (float *)(A_tile_B_tile + A_tile_offset/sizeof(float));

    int bx = blockIdx.x; 
    int by = blockIdx.y; 
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Identify the row and column of the output matrix C. 
    int row = by * tile_size + ty;
    int colStart = bx * tile_size * COARSE_FACTOR + tx; // each thread block computes COARSE_FACTOR columns of C. 

    // Initialize the sum_value to all output elements. 
    float sum_val[COARSE_FACTOR];
    for (int i = 0; i < COARSE_FACTOR; i++) {
        sum_val[i] = 0.0f;
    }

    // Loop over the A and B tiles.
    for (int ph = 0; ph < ceil(K / (float)tile_size); ++ph) {
        if ((row < J) && (ph * tile_size + tx < K)) {
            A_tile[ty * tile_size + tx] = A[row * K + ph * tile_size + tx]; // L is the height and K is the width of the matrix A.
        } else {
            A_tile[ty * tile_size + tx] = 0.0f;
        }

        for (int c = 0; c < COARSE_FACTOR; c++) {
            int col = colStart + c * tile_size;
            if ((ph * tile_size + ty < K) && (col < L)) {
                B_tile[ty * tile_size + tx] = B[(ph * tile_size + ty) * L + col];
            } else {
                B_tile[ty * tile_size + tx] = 0.0f;
            }

            __syncthreads();

            for (int k = 0; k < tile_size; k++) {
                sum_val[c] += A_tile[ty * tile_size + k] * B_tile[k * tile_size + tx];
            }

            __syncthreads();
        }
    }

    for (int c = 0; c < COARSE_FACTOR; c++) {
        int col = colStart + c * tile_size;
        if ((row < J) && (col < L)) {
            C[row * L + col] = sum_val[c];
        }
    }   
}

void coarseMatrixMul(int J, int K, int L, float *A, float *B, float *C) {
    int size_A = J * K * sizeof(float);
    int size_B = K * L * sizeof(float);
    int size_C = J * L * sizeof(float);

    float *A_d, *B_d, *C_d;
    
    // Determine the appropriate tile size. 
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0); // 0 means the first GPU.
    int tile_size = calculate_appropriate_tile_size(device_prop);

    // Part 1: Allocate device memory for A, B, and C. 
    // copy A and B to device memory. 
    hipError_t err = hipMalloc((void**)&A_d, size_A);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    
    err = hipMalloc((void**)&B_d, size_B);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    err = hipMalloc((void**)&C_d, size_C);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    hipMemcpy(A_d, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size_B, hipMemcpyHostToDevice);
    
    // Part 2: Launch the kernel to perform the matrix multiplication. 
    dim3 dimBlock(tile_size, tile_size, 1);
    dim3 dimGrid(ceil(J / (float)tile_size), ceil(L / (float)tile_size), 1);
    size_t size = 2 * tile_size * tile_size * sizeof(float); // 2 tiles for A and B.
    coarseMatrixMul_kernel<<<dimGrid, dimBlock, size>>>(J, K, L, A_d, B_d, C_d, tile_size, size/2, size/2);

    // Part 3: Copy the result back to the host. 
    // free the device memory. 
    hipMemcpy(C, C_d, size_C, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main() {
    int J = 3; // number of rows in matrix A and rows in matrix C.
    int K = 5; // number of columns in matrix A and rows in matrix B.
    int L = 4; // number of columns in matrix B and columns in matrix C.

    int size_A = J * K * sizeof(float);
    int size_B = K * L * sizeof(float);
    int size_C = J * L * sizeof(float);

    float *A = (float *)malloc(size_A);
    float *B = (float *)malloc(size_B);
    float *C = (float *)malloc(size_C);

    // Initialize the matrices A and B with random values. 
    for (int i = 0; i < J; i++) {
        for (int j = 0; j < K; j++) {
            A[i * K + j] = rand() % 2;
        }
    }

    for (int i = 0; i < K; i++) {
        for (int j = 0; j < L; j++) {
            B[i * L + j] = rand() % 2;
        }
    }
    
    // Print the matrices A and B. 
    printMatrix(A, K, J);
    printMatrix(B, L, K);

    // Call the matrix multiplication function. 
    coarseMatrixMul(J, K, L, A, B, C);

    // Print the result. 
    printMatrix(C, L, J);

    // Free the memory allocated for the matrices. 
    free(A);
    free(B);
    free(C);

    return 0;
}