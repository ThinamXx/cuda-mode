#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define NUM_BUCKETS 7
#define COARSENING_FACTOR 3

__global__ void histogramKernelCoarsening(char *data, unsigned int *histogram, unsigned int N) {
    // Initialize the privatized bins. 
    __shared__ unsigned int hist_s[NUM_BUCKETS];
    for (unsigned int bin = threadIdx.x; bin < NUM_BUCKETS; bin += blockDim.x) {
        hist_s[bin] = 0u;
    }

    __syncthreads();

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int i = tid * COARSENING_FACTOR; i < min(N, (tid + 1) * COARSENING_FACTOR); i++) {
        int alpha_index = data[i] - 'a';
        if (alpha_index >= 0 && alpha_index < 26) {
            atomicAdd(&(hist_s[alpha_index / 4]), 1);
        }
    }

    __syncthreads();

    for (unsigned int bin = threadIdx.x; bin < NUM_BUCKETS; bin += blockDim.x) {
        unsigned int binValue = hist_s[bin];
        if (binValue >0) {
            atomicAdd(&(histogram[bin]), binValue);
        }
    }
}

__global__ void histogramKernelInterleaved(char *data, unsigned int *histogram, unsigned int N) {
    // Initialize the privatized bins. 
    __shared__ unsigned int hist_s[NUM_BUCKETS];
    for (unsigned int bin = threadIdx.x; bin < NUM_BUCKETS; bin += blockDim.x) {
        hist_s[bin] = 0u;
    }

    __syncthreads();

    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (unsigned int i = tid; i < N; i += blockDim.x * gridDim.x) {
        int alpha_index = data[i] - 'a';
        if (alpha_index >= 0 && alpha_index < 26) {
            atomicAdd(&(hist_s[alpha_index / 4]), 1);
        }
    }

    __syncthreads();

    for (unsigned int bin = threadIdx.x; bin < NUM_BUCKETS; bin += blockDim.x) {
        unsigned int binValue = hist_s[bin];
        if (binValue >0) {
            atomicAdd(&(histogram[bin]), binValue);
        }
    }
}

void histogramCuda(char *data, unsigned int *histogram, unsigned int *histogram_interleaved, unsigned int N) {
    int size = N * sizeof(char);

    int size_histogram = NUM_BUCKETS * sizeof(unsigned int);

    char *d_data;
    unsigned int *d_histogram;
    unsigned int *d_histogram_interleaved;

    // 1. Allocate device memory for the input and output arrays.
    hipError_t err = hipMalloc((void**)&d_data, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&d_histogram, size_histogram);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&d_histogram_interleaved, size_histogram);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    // 2. Copy the input data to the device.
    err = hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);

    // 3. Set 0 to the histogram.
    err = hipMemset(d_histogram, 0, size_histogram);
    err = hipMemset(d_histogram_interleaved, 0, size_histogram);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    // 4. Launch the kernel.
    dim3 dimBlock(1024, 1, 1);
    dim3 dimGrid(ceil(N / 1024.0));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    histogramKernelCoarsening<<<dimGrid, dimBlock>>>(d_data, d_histogram, N);
    histogramKernelInterleaved<<<dimGrid, dimBlock>>>(d_data, d_histogram_interleaved, N);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // 5. Copy the output data to the host.
    hipMemcpy(histogram, d_histogram, size_histogram, hipMemcpyDeviceToHost);
    hipMemcpy(histogram_interleaved, d_histogram_interleaved, size_histogram, hipMemcpyDeviceToHost);

    // 6. Free the device memory.
    hipFree(d_data);
    hipFree(d_histogram);
    hipFree(d_histogram_interleaved);
    printf("Time taken: %f ms\n", milliseconds);
}

int main() {
    char data[] = "programming massively parallel processors.";
    unsigned int N = strlen(data);

    // We will use 7 buckets because 26 / 4. 
    unsigned int *histogram_cuda = (unsigned int *)malloc(7 * sizeof(unsigned int));
    unsigned int *histogram_cuda_interleaved = (unsigned int *)malloc(7 * sizeof(unsigned int));
    printf("Input: %s\n", data);

    histogramCuda(data, histogram_cuda, histogram_cuda_interleaved, N);
    printf("Histogram by letter groups:\n");
    for (int i = 0; i < 7; i++) {
        printf("Group %d (%c-%c): %d %d\n",
            i, 
            'a' + (i * 4), 
            'a' + (i * 4 + 3 < 26 ? i * 4 + 3 : 25), 
            histogram_cuda[i],
            histogram_cuda_interleaved[i]);
    }

    free(histogram_cuda);
    
    return 0;
}