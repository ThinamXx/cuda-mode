#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define NUM_BUCKETS 7

void histogramSeq(char *data, unsigned int *histogram, unsigned int N) {
    for (int i = 0; i < N; i++) {
        int index = data[i] - 'a';
        if (index >= 0 && index < 26) {
            histogram[index / 4]++;
        }
    }
}

__global__ void histogramKernel(char *data, unsigned int *histogram, unsigned int N) {
    // This kernel demonstrates the use of atomic operations (atomicAdd) to update the histogram.
    // Atomic operations are used to ensure that only one thread can update the memory location at a time.
    // This is important when multiple threads are updating the same location in the histogram.

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        int alpha_index = data[tid] - 'a';
        if (alpha_index >= 0 && alpha_index < 26) {
            atomicAdd(&(histogram[alpha_index / 4]), 1);
        }
    }
}

void histogramCuda(char *data, unsigned int *histogram, unsigned int N) {
    int size = N * sizeof(char);
    int size_histogram = 7 * sizeof(unsigned int); // 7 buckets with 4 letters each for a-z. 

    char *d_data;
    unsigned int *d_histogram;
    
    // 1. Allocate device memory for the input and output arrays.
    hipError_t err = hipMalloc((void**)&d_data, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&d_histogram, size_histogram);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    // 2. Copy the input data to the device.
    err = hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);

    // 3. Launch the kernel.
    dim3 block(1024);
    dim3 grid(ceil(N / (float)block.x));

    histogramKernel<<<grid, block>>>(d_data, d_histogram, N);

    // 4. Copy the output data to the host.
    err = hipMemcpy(histogram, d_histogram, size_histogram, hipMemcpyDeviceToHost);

    // 5. Free the device memory.
    hipFree(d_data);
    hipFree(d_histogram);
    
}

__global__ void histogramKernelPrivate(char *data, unsigned int *histogram, unsigned int N) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N) {
        int alpha_index = data[tid] - 'a';
        if (alpha_index >= 0 && alpha_index < 26) {
            atomicAdd(&(histogram[blockIdx.x * NUM_BUCKETS + alpha_index / 4]), 1);
        }
    }

    if (blockIdx.x > 0) {
        __syncthreads();

        for (unsigned int bin = threadIdx.x; bin < NUM_BUCKETS; bin += blockDim.x) {
            unsigned int binValue = histogram[blockIdx.x * NUM_BUCKETS + bin];
            if (binValue >0) {
                atomicAdd(&(histogram[bin]), binValue);
            }
        }
    }
}

void histogramCudaPrivate(char *data, unsigned int *histogram, unsigned int N) {
    int size = N * sizeof(char);

    // Setup the grid and block dimensions.
    int threads_per_block = 1024;
    int blocks_per_grid = ceil(N / (float)threads_per_block);

    int size_histogram = NUM_BUCKETS * blocks_per_grid * sizeof(unsigned int);

    char *d_data;
    unsigned int *d_histogram;

    // 1. Allocate device memory for the input and output arrays.
    hipError_t err = hipMalloc((void**)&d_data, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&d_histogram, size_histogram);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    // 2. Copy the input data to the device.
    err = hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);

    // 3. Set 0 to the histogram.
    err = hipMemset(d_histogram, 0, size_histogram);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    // 4. Launch the kernel.
    dim3 dimBlock(threads_per_block);
    dim3 dimGrid(blocks_per_grid);

    histogramKernelPrivate<<<dimGrid, dimBlock>>>(d_data, d_histogram, N);

    // 5. Copy the output data to the host.
    hipMemcpy(histogram, d_histogram, size_histogram, hipMemcpyDeviceToHost);

    // 6. Free the device memory.
    hipFree(d_data);
    hipFree(d_histogram);
}

int main() {
    char data[] = "programming massively parallel processors.";
    unsigned int N = strlen(data);

    // We will use 7 buckets because 26 / 4. 
    unsigned int *histogram_seq = (unsigned int *)malloc(7 * sizeof(unsigned int));
    unsigned int *histogram_cuda = (unsigned int *)malloc(7 * sizeof(unsigned int));
    unsigned int *histogram_cuda_private = (unsigned int *)malloc(7 * sizeof(unsigned int));
    printf("Input: %s\n", data);

    histogramSeq(data, histogram_seq, N);
    histogramCuda(data, histogram_cuda, N);
    histogramCudaPrivate(data, histogram_cuda_private, N);

    printf("Histogram by letter groups:\n");
    for (int i = 0; i < 7; i++) {
        printf("Group %d (%c-%c): %d %d %d\n",
            i, 
            'a' + (i * 4), 
            'a' + (i * 4 + 3 < 26 ? i * 4 + 3 : 25), 
            histogram_seq[i],
            histogram_cuda[i],
            histogram_cuda_private[i]);
    }

    free(histogram_seq);
    free(histogram_cuda);
    free(histogram_cuda_private);
    
    return 0;
}