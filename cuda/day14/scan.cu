#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define SECTION_SIZE 32

void sequentialScanCPU(float *input, float *output, int N) {
    output[0] = input[0];
    for (int idx = 1; idx < N; ++idx) {
        output[idx] = output[idx - 1] + input[idx];
    }
}

__global__ void scanKernel(float *input, float *output, int N) {
    __shared__ float XY[SECTION_SIZE];
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 

    if (idx < N) {
        XY[threadIdx.x] = input[idx];
    } else {
        XY[threadIdx.x] = 0.0f;
    }
    __syncthreads();

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        float temp;

        if (threadIdx.x >= stride) {
            temp = XY[threadIdx.x] + XY[threadIdx.x - stride];
        }
        __syncthreads();

        if (threadIdx.x >= stride) {
            XY[threadIdx.x] = temp;
        }
        __syncthreads();
    }

    if (idx < N) {
        output[idx] = XY[threadIdx.x];
    }
}

void sequentialScan(float *input, float *output, int N) {
    int size = N * sizeof(float);

    float *d_input, *d_output;

    // 1. Allocate device memory for the input and output arrays.
    hipError_t err = hipMalloc((void**)&d_input, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&d_output, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    // 2. Copy the input array to the device.
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    // 3. Launch the kernel to perform the scan.
    dim3 dimGrid(ceil(N / (float)SECTION_SIZE), 1, 1);
    dim3 dimBlock(SECTION_SIZE, 1, 1);
    scanKernel<<<dimGrid, dimBlock>>>(d_input, d_output, N);

    // 4. Copy the result from the device to the host.
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    // 5. Free the device memory.
    hipFree(d_input);
    hipFree(d_output);
}

int main() {
    int N = 7;
    float input[N] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0};
    float output[N];
    float output_cpu[N];

    sequentialScanCPU(input, output_cpu, N);
    sequentialScan(input, output, N);

    printf("Sequential scan result: ");
    for (int i = 0; i < N; i++) {
        printf("%f ", output_cpu[i]);
    }
    printf("\n");

    printf("Parallel scan result: ");
    for (int i = 0; i < N; i++) {
        printf("%f ", output[i]);
    }
    printf("\n");

    return 0;
}