#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_DIM 4

__global__ void sumReductionSegmentShared(float *input, float *output) {
    __shared__ float shared_data[BLOCK_DIM];

    // Each block will process one segment and each block will 
    // process 2 * BLOCK_DIM elements.
    unsigned int segment = 2 * BLOCK_DIM * blockIdx.x; 

    unsigned int tid = threadIdx.x;
    unsigned int segment_tid = segment + threadIdx.x;

    shared_data[tid] = input[segment_tid] + input[segment_tid + BLOCK_DIM];
    __syncthreads();

    for (unsigned int stride = BLOCK_DIM / 2; stride >= 1; stride /= 2) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }

        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(output, shared_data[0]);
    }
}


void sumReduction(float *input, float *output_optimized, int N) {
    int size = N * sizeof(float);

    float *d_input_optimized, *d_output_optimized;

    // 1. Allocate device memory for the input and output arrays.
    hipError_t err = hipMalloc((void**)&d_input_optimized, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&d_output_optimized, sizeof(float));
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    // 2. Copy the input array to the device memory.
    hipMemcpy(d_input_optimized, input, size, hipMemcpyHostToDevice);

    // 3. Launch the kernel. 
    dim3 block(BLOCK_DIM, 1, 1);
    dim3 grid(ceil(N / float(2 * BLOCK_DIM)), 1, 1);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    sumReductionSegmentShared<<<grid, block>>>(d_input_optimized, d_output_optimized);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // 4. Copy the output array to the host memory.
    hipMemcpy(output_optimized, d_output_optimized, sizeof(float), hipMemcpyDeviceToHost);

    // 5. Free the device memory.
    hipFree(d_input_optimized);
    hipFree(d_output_optimized);
    printf("GPU time taken: %f milliseconds\n", milliseconds);
}

int main() {
    int N = 8;
    float input[N] = {4.0, 7.0, 2.0, 3.0, 8.0, 5.0, 9.0, 6.0};
    
    float *output_optimized = (float *)malloc(sizeof(float));

    sumReduction(input, output_optimized, N);

    printf("Sum of the input optimized: %f\n", output_optimized[0]);

    free(output_optimized);
    return 0;
}