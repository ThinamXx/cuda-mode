#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void sumReductionKernel(float *input, float *output, int N) {
    unsigned int tid = threadIdx.x;

    for (unsigned int stride = 1; stride < N; stride *= 2) {
        int index = 2 * stride * tid; // tid is the offset. 
        if (index + stride < N) {
            input[index] += input[index + stride];
        }

        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *output = input[0];
    }
}

__global__ void sumReductionKernelOptimized(float *input, float *output, int N) {
    unsigned int tid = threadIdx.x;

    for (unsigned int stride = N / float(2); stride >= 1; stride /= 2) {
        if (threadIdx.x < stride) {
            input[tid] += input[tid + stride];
        }

        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *output = input[0];
    }
}

void sumReduction(float *input, float *output, float *output_optimized, int N) {
    int size = N * sizeof(float);

    float *d_input, *d_input_optimized, *d_output, *d_output_optimized;

    // 1. Allocate device memory for the input and output arrays.
    hipError_t err = hipMalloc((void**)&d_input, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&d_input_optimized, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&d_output, sizeof(float));
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&d_output_optimized, sizeof(float));
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    // 2. Copy the input array to the device memory.
    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);
    hipMemcpy(d_input_optimized, input, size, hipMemcpyHostToDevice);

    // 3. Launch the kernel. 
    dim3 block(N / 2, 1, 1);
    dim3 grid(1, 1, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    sumReductionKernel<<<grid, block>>>(d_input, d_output, N);
    sumReductionKernelOptimized<<<grid, block>>>(d_input_optimized, d_output_optimized, N);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // 4. Copy the output array to the host memory.
    hipMemcpy(output, d_output, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(output_optimized, d_output_optimized, sizeof(float), hipMemcpyDeviceToHost);

    // 5. Free the device memory.
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_output_optimized);
    printf("GPU time taken: %f milliseconds\n", milliseconds);
}

int main() {
    // Since, reduction requires collaboration between threads, 
    // we will use 1 block of threads. The max number of threads 
    // in a block is 1024, means that we can process 2*1024 elements. 
    int N = 8;
    float input[N] = {4.0, 7.0, 2.0, 3.0, 8.0, 5.0, 9.0, 6.0};
    
    float *output = (float *)malloc(sizeof(float));
    float *output_optimized = (float *)malloc(sizeof(float));

    sumReduction(input, output, output_optimized, N);

    printf("Sum of the input: %f\n", output[0]);
    printf("Sum of the input optimized: %f\n", output_optimized[0]);

    free(output);
    free(output_optimized);
    return 0;
}