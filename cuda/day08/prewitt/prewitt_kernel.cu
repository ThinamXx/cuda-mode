#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#define FILTER_RADIUS 1
#define FILTER_SIZE (2 * FILTER_RADIUS + 1)
#define TILE_DIM 32

__constant__ float PREWITT_X[FILTER_SIZE][FILTER_SIZE];
__constant__ float PREWITT_Y[FILTER_SIZE][FILTER_SIZE];

__global__ void prewitt_kernel(
    float *image,
    float *output,
    int width,
    int height
) {
    __shared__ float input_tile[TILE_DIM][TILE_DIM];

    int row = blockIdx.y * TILE_DIM + threadIdx.y;
    int col = blockIdx.x * TILE_DIM + threadIdx.x;

    if (row < height && col < width) {
        input_tile[threadIdx.y][threadIdx.x] = image[row * width + col];
    } else {
        input_tile[threadIdx.y][threadIdx.x] = 0.0f;
    }

    __syncthreads();

    if (row < height && col < width) {
        float sum_x = 0.0f;
        float sum_y = 0.0f;

        for (int f_row = 0; f_row < FILTER_SIZE; f_row++) {
            for (int f_col = 0; f_col < FILTER_SIZE; f_col++) {
                if ((int)threadIdx.x - FILTER_RADIUS + f_col >= 0 &&
                    (int)threadIdx.x - FILTER_RADIUS + f_col < TILE_DIM &&
                    (int)threadIdx.y - FILTER_RADIUS + f_row >= 0 &&
                    (int)threadIdx.y - FILTER_RADIUS + f_row < TILE_DIM) {
                    sum_x += input_tile[threadIdx.y + f_row][threadIdx.x + f_col] * PREWITT_X[f_row][f_col];
                    sum_y += input_tile[threadIdx.y + f_row][threadIdx.x + f_col] * PREWITT_Y[f_row][f_col];
                } 
                else {
                    if (row - FILTER_RADIUS + f_row >= 0 &&
                        row - FILTER_RADIUS + f_row < height &&
                        col - FILTER_RADIUS + f_col >= 0 &&
                        col - FILTER_RADIUS + f_col < width) {
                        sum_x += image[(row - FILTER_RADIUS + f_row) * width + (col - FILTER_RADIUS + f_col)] * PREWITT_X[f_row][f_col];
                        sum_y += image[(row - FILTER_RADIUS + f_row) * width + (col - FILTER_RADIUS + f_col)] * PREWITT_Y[f_row][f_col];
                    }
                }
            }
        }

        output[row * width + col] = sqrtf(sum_x * sum_x + sum_y * sum_y);
    }
}

void prewitt_filters() {
    float prewitt_x[FILTER_SIZE][FILTER_SIZE] = {
        {-1, 0, 1},
        {-1, 0, 1},
        {-1, 0, 1}
    };

    float prewitt_y[FILTER_SIZE][FILTER_SIZE] = {
        {-1, -1, -1},
        { 0,  0,  0},
        { 1,  1,  1}
    };

    hipMemcpyToSymbol(HIP_SYMBOL(PREWITT_X), prewitt_x, FILTER_SIZE * FILTER_SIZE * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(PREWITT_Y), prewitt_y, FILTER_SIZE * FILTER_SIZE * sizeof(float));
}

torch::Tensor prewitt_cuda_forward(torch::Tensor input) {
    input = input.contiguous();

    const int height = input.size(0);
    const int width = input.size(1);

    auto output = torch::zeros_like(input);

    const dim3 dimBlock(TILE_DIM, TILE_DIM, 1);
    const dim3 dimGrid(ceil(width / (float)TILE_DIM), ceil(height / (float)TILE_DIM), 1);

    prewitt_filters();

    prewitt_kernel<<<dimGrid, dimBlock>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        width,
        height
    );

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Error in prewitt_cuda_forward: %s\n", hipGetErrorString(err));
    }

    return output;
}