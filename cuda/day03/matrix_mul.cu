
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Create a function to print the matrix. 
void printMatrix(float *matrix, int width, int height) {
    printf("\n\n");

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            printf("%f ", matrix[i * width + j]);
        }
        printf("\n");
    }

    printf("\n\n");
}

__global__ void matrixMul_kernel(int N, float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < N) && (col < N)) {
        float c_sum = 0.0f;
        // Perform the matrix multiplication
        // using the row-major order. 
        for (int k = 0; k < N; k++) {
            c_sum += A[row * N + k] * B[k * N + col];
        }

        C[row * N + col] = c_sum;
    }    
}

void matrixMul(int N, float *A, float *B, float *C) {
    int size = N * N * sizeof(float);
    float *A_d, *B_d, *C_d;

    // Part 1: Allocate device memory for A, B, and C.
    // copy A and B to device memory. 
    hipError_t err = hipMalloc((void**)&A_d, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&B_d, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&C_d, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    
    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

    // Part 2: Call the kernel to launch the grid of threads. 
    // to perform the matrix multiplication. 
    dim3 dimGrid(ceil(N / 32.0), ceil(N / 32.0), 1);
    dim3 dimBlock(32, 32, 1);
    matrixMul_kernel<<<dimGrid, dimBlock>>>(N, A_d, B_d, C_d);

    // Part 3: Copy the result back to the host. 
    // free the device memory. 
    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main() {
    int N = 5;
    
    int size = N * N * sizeof(float);

    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C = (float *)malloc(size);

    // Initialize the matrices. 
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int offset = i * N + j;
            A[offset] = rand() % 2;
            B[offset] = rand() % 2;
        }
    }

    // Print the matrices. 
    printMatrix(A, N, N);
    printMatrix(B, N, N);

    // Call the matrix multiplication function. 
    matrixMul(N, A, B, C);

    // Print the result. 
    printMatrix(C, N, N);

    // Free the allocated memory. 
    free(A);
    free(B);
    free(C);
    return 0;
}

