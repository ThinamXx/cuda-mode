#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define c0 0.00f
#define c1 0.50f
#define c2 0.87f
#define c3 1.00f
#define c4 0.87f
#define c5 0.50f
#define c6 0.00f

void printMatrix(float *matrix, int width, int height, int depth) {
    printf("\n\n");

    for (int z = 0; z < depth; z++) {
        for (int x = 0; x < height; x++) {
            for (int y = 0; y < width; y++) {
                printf("%f ", matrix[z * width * height + x * width + y]);
            }
            printf("\n");
        }
        printf("\n");
    }

    printf("\n\n");
}

__global__ void stencilKernel(float *in, float *out, int N) {
    unsigned int i = blockIdx.z * blockDim.z + threadIdx.z;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= 1 && i < N - 1 && j >= 1 && j < N - 1 && k >= 1 && k < N - 1) {
        out[i * N * N + j * N + k] = c0 * in[i * N * N + j * N + k] +
                                     c1 * in[i * N * N + j * N + (k - 1)] + 
                                     c2 * in[i * N * N + j * N + (k + 1)] + 
                                     c3 * in[i * N * N + (j - 1) * N + k] + 
                                     c4 * in[i * N * N + (j + 1) * N + k] + 
                                     c5 * in[(i - 1) * N * N + j * N + k] + 
                                     c6 * in[(i + 1) * N * N + j * N + k];
    }
}

void stencilSweep(float *in, float *out, int N) {
    int size = N * N * N * sizeof(int);
    float *in_d, *out_d;

    // Part 1: Allocate device memory for input and output.
    hipError_t err = hipMalloc((void**)&in_d, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }
    err = hipMalloc((void**)&out_d, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    }

    // Part 2: Copy input to the device memory.
    hipMemcpy(in_d, in, size, hipMemcpyHostToDevice);

    // Part 3: Launch the kernel.
    dim3 dimBlock(8, 8, 8);
    dim3 dimGrid(ceil(N / 8.0), ceil(N / 8.0), ceil(N / 8.0));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    stencilKernel<<<dimGrid, dimBlock>>>(in_d, out_d, N);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Part 4: Copy output back to the host memory.
    hipMemcpy(out, out_d, size, hipMemcpyDeviceToHost);

    // Part 5: Free the device memory.
    hipFree(in_d);
    hipFree(out_d);

    printf("GPU time taken: %f milliseconds\n", milliseconds);
}

int main() {
    int N = 4;

    int size = N * N * N * sizeof(int);
    float *in = (float *)malloc(size);
    float *out = (float *)malloc(size);

    for (int i = 0; i < N * N * N; i++) {
        in[i] = rand() % 2;
    }

    printMatrix(in, N, N, N);

    stencilSweep(in, out, N);

    printMatrix(out, N, N, N);

    free(in);
    free(out);

    return 0;
}